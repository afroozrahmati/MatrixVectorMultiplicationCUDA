#include "hip/hip_runtime.h"
﻿/*
 * CSS-535 Lab 03: CUDA GEMV Implementation
 * Authors: Afrooz Rahmati & Tony Varela
 *
 * Description: This is my (Tony) reimplementation of Afrooz's original code in C++ (thanks to her for getting this started!).
 * For now, let's focus on Part 0 - the naive implementation of GEMV using CUDA.
 */

// included header files 

// CUDA stuff 
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

#include <random> // for random initialization
#include <chrono> // timing
#include <iostream> // for output
#include <string>

/*
 * With shared memory, we want to take advantage of locality.
 * The only reasonable thing to share, imo, among these threads is the input vector x.
 * Ideally, if each kernel/thread is only in charge of one element, then my only worry is to make sure that
 * the input vector x is shared AMAP amongst the threads of a block, so they dont have to pull it from global memory...
 * Since we're basically testing only 10000 elements at this point, that'd be around 40KB of data, which should fit into the default
 * configured shared memory size.
 *
 * But, I wouldnt want just ONE kernel loading everything... I need each kernel to load a fair portion of the vector into shared memory.
 * Which basically just ends up being another same scheme as what i already had for gemv itself...
 */
__global__ void shared_gemv(const float *A, const float *x, float *y, const size_t M, const size_t N) {
	const size_t total_thread_num{ static_cast<size_t>(gridDim.x) * blockDim.x };
	const size_t tid{ threadIdx.x + static_cast<size_t>(blockIdx.x) * blockDim.x };
	
	extern __shared__ float x_shared[]; // extern shared because we dont know the shared memory size at compile time 

	// remember, shared memory is PER BLOCK.

	// let's get the TPB count - threads per block, which should just be blockDim.x
	const size_t tpb{ blockDim.x };
	const size_t tidx{ threadIdx.x };
	size_t x_stride{ N / tpb };
	if (x_stride == 0) {
		if (tidx >= N) return; // i think this means... if the current thread within the block is greater than N, then dont bother loading anything in.
		x_stride = 1;
	}
	const size_t x_begin_index{tidx*x_stride};
	size_t x_end_index{ x_begin_index + x_stride };
	x_end_index += (tidx == static_cast<size_t>(tpb) - 1) // if you are the last thread within the block...
		? ((N <= tpb) ? 0 : N % tpb)
		: 0;

	// printf("THIS KERNEL IS LOADING FROM %llu TO %llu from X\n", x_begin_index, x_end_index);

	// this is the ultimate goal of what I want before proceeding to the rest of the calculating 
	for (size_t i{ x_begin_index }; i < x_end_index; i++) {
		x_shared[i] = x[i];
	}

	__syncthreads(); // make sure all of the data is loaded before proceeding 
	

	size_t stride{M / total_thread_num};

	if (stride == 0) {
		if (tid >= M) return;
		stride = 1;
	}
	// else, that means stride >= 1 (more elements than threads); if the current thread index is the LAST ONE, we need to consider the possible remainders. and ONLY IF we have more vector elements than threads.
	const size_t begin_index{tid * stride};
	size_t end_index{begin_index + stride};
	end_index += (tid == static_cast<size_t>(total_thread_num) - 1)
		             ? ((M <= total_thread_num) ? 0 : M % total_thread_num)
		             : 0;
	for (size_t i{begin_index}; i < end_index; i++) {
		/*y[i] = 0.0f;*/
		float sum{0.0f};
		for (size_t j{0}; j < N; j+=8) {
			sum += A[i * M + j] * x[j];
			sum += A[i * M + j + 1] * x_shared[j + 1];
			sum += A[i * M + j + 2] * x_shared[j + 2];
			sum += A[i * M + j + 3] * x_shared[j + 3];
			sum += A[i * M + j + 4] * x_shared[j + 4];
			sum += A[i * M + j + 5] * x_shared[j + 5];
			sum += A[i * M + j + 6] * x_shared[j + 6];
			sum += A[i * M + j + 7] * x_shared[j + 7];
		}
		y[i] = sum;
	}
}



//__global__ void mat_mul_tiled(float* vec, float* mat, float* res, const int N) {
//
//    __shared__  float As[BLOCK_SIZE][BLOCK_SIZE];
//    __shared__  float Bs[BLOCK_SIZE];
//	
//    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y; //the row index of As and Bs
//    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x; //the column index of As and Bs
//    float tmp = 0;
//    int idx;
//
//
//    for (int i = 0; i < gridDim.y; ++i) //initialize the ID index
//    {
//		
//        idx = row * N + i * BLOCK_SIZE + threadIdx.x;
//		
//        //if N is not divisible by block width
//        if (idx >= N*N)
//        {
//            As[threadIdx.y][threadIdx.x] = 0; 
//        }
//        else{
//            As[threadIdx.y][threadIdx.x] = mat[idx];   
//        }
//
//        idx = col + i * BLOCK_SIZE;
//
//
//        //if N is not divisible by block width
//        if(idx >= N)  
//        {
//            Bs[threadIdx.x] = 0;
//        }
//        else{
//            Bs[threadIdx.x] = vec[idx];  
//        }
//
//		
//
//        //Matrix and vectors should be loaded completely before any further process
//        __syncthreads();
//
//        //multiply sub matrices
//        for (int k = 0; k < BLOCK_SIZE; k++) 
//        {
//            tmp += As[threadIdx.y][k] * Bs[k];
//			
//        }
//
//        __syncthreads();
//    }
//
//    //write result back to global memory
//    if(row < N && col < N)
//    {
//        res[row] = tmp;  
//    }
//}

// Credits to Brian Luger for the main structure of this program (just the way it is divided, I learned this from our time together on Lab 2)
int main(int argc, char **argv) {
	// TODO: create command line arguments to configure grid/block dimensions
	// This program should only take in the M and N dimensions; within the program, we figure out the execution configurations ourselves
	if (argc != 4) {
		std::cout << "Input: Vector_Size GridDim.x BlockDim.x. Exiting...\n";
		return -1;
	}

	// for now, let's put the matrix/vector dimensions in here as well
	const size_t M{std::stoul(std::string{argv[1]})};
	const size_t N{std::stoul(std::string{argv[1]})};
	// let's create the grid / block configuration, but just really simply.
	dim3 grid{std::stoul(std::string{argv[2]})}; // (1, 1, 1)
	dim3 block{std::stoul(std::string{argv[3]})};

	// yes, I know they're always going to be square, but I like separating M and N for my own understanding.
	// TODO: consider experimenting with thrust device/host vectors as well

	// seed RNG
	std::default_random_engine dre;
	dre.seed(3); // seeded for reproducibility
	const std::uniform_real_distribution<float> uniform_dist(-10, 10); // uniform distribution [-10, 10]

	// hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	
	// allocate host memory
	float *m{new float[M * N]};
	float *v_in{new float[N]};
	float *v_out_naive{new float[M]};
	float *v_out_cublas{new float[M]};

	// allocate device memory
	float *d_m, *d_v_in, *d_v_out_naive, *d_v_out_cublas;
	std::cout << "ERROR CODE: " << hipGetErrorString(hipMalloc(reinterpret_cast<void**>(&d_m), sizeof(float) * M * N))
		<< std::endl;
	std::cout << "ERROR CODE: " << hipGetErrorString(hipMalloc(reinterpret_cast<void**>(&d_v_in), sizeof(float) * N))
		<< std::endl;
	std::cout << "ERROR CODE: " << hipGetErrorString(
		hipMalloc(reinterpret_cast<void**>(&d_v_out_naive), sizeof(float) * M)) << std::endl;
	std::cout << "ERROR CODE: " << hipGetErrorString(
		hipMalloc(reinterpret_cast<void**>(&d_v_out_cublas), sizeof(float) * M)) << std::endl;
	// initialize host array with random data

	// for the matrix 
	for (size_t i{ 0 }; i < M; i++) for (size_t j{ 0 }; j < N; j++) m[i * M + j] = (i + j) % 2 + 1;//uniform_dist(dre);
	for (size_t i{ 0 }; i < N; i++) v_in[i] = (i % 2) + 1;//uniform_dist(dre);

	// copy m and v_in into device memory, time it as well
	auto d2h_start = std::chrono::high_resolution_clock::now();
	std::cout << "ERROR CODE: " << hipGetErrorString(hipMemcpy(d_m, m, sizeof(float) * M * N, hipMemcpyHostToDevice))
		<< std::endl;
	std::cout << "ERROR CODE: " << hipGetErrorString(
		hipMemcpy(d_v_in, v_in, sizeof(float) * N, hipMemcpyHostToDevice)) << std::endl;
	auto d2h_end = std::chrono::high_resolution_clock::now();
	auto d2h_duration = std::chrono::duration_cast<std::chrono::microseconds>(d2h_end - d2h_start).count();

	auto naive_exec_start = std::chrono::high_resolution_clock::now();
	shared_gemv << <grid, block, N*sizeof(float) >> >(d_m, d_v_in, d_v_out_naive, M, N);
	std::cout << "ERROR CODE: " << hipGetErrorString(hipDeviceSynchronize()) << std::endl;
	// since the kernels are executed asynchronously, need to sync so that we can get accurate timing
	auto naive_exec_end = std::chrono::high_resolution_clock::now();
	auto naive_exec_duration = std::chrono::duration_cast<std::chrono::microseconds>(naive_exec_end - naive_exec_start).
		count();

	// copy d_v_out_naive back into host
	auto h2d_start = std::chrono::high_resolution_clock::now();
	hipMemcpy(v_out_naive, d_v_out_naive, sizeof(float) * M, hipMemcpyDeviceToHost);
	auto h2d_end = std::chrono::high_resolution_clock::now();
	auto h2d_duration = std::chrono::duration_cast<std::chrono::microseconds>(h2d_end - h2d_start).count();

	// get total inclusive time 
	auto gpu_transfer_total_duration = h2d_duration + d2h_duration;

	// cublas stuff
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);
	float a{1.0f};
	float b{0.0f};
	auto cublas_exec_start = std::chrono::high_resolution_clock::now();
	/*naive_gemv << <grid, block >> > (d_m, d_v_in, d_v_out_naive, M, N);*/
	hipblasSgemv(cublas_handle, HIPBLAS_OP_T, N, M, &a, d_m, N, d_v_in, 1, &b, d_v_out_cublas, 1);
	// since the kernels are executed asynchronously, need to sync so that we can get accurate timing
	auto cublas_exec_end = std::chrono::high_resolution_clock::now();
	auto cublas_exec_duration = std::chrono::duration_cast<std::chrono::microseconds>(
			cublas_exec_end - cublas_exec_start).
		count();

	// copy d_v_out_naive back into host
	hipMemcpy(v_out_cublas, d_v_out_cublas, sizeof(float) * M, hipMemcpyDeviceToHost);

	float rse{0.0f};
	for (size_t i{0}; i < M; i++) rse += abs(v_out_cublas[i] - v_out_naive[i]);

	std::cout << "RSE: " << rse << std::endl;

	std::cout <<
		"Total Inclusive Time, Naive Execution Time, Naive Total Time\n";
	std::cout << gpu_transfer_total_duration << ", " << naive_exec_duration << ", " <<
		naive_exec_duration +
		gpu_transfer_total_duration << '\n';

	// clean up
	hipblasDestroy(cublas_handle);
	hipFree(d_v_out_cublas);
	hipFree(d_v_out_naive);
	hipFree(d_v_in);
	hipFree(d_m);

	delete[] v_out_naive;
	delete[] v_in;
	delete[] m;

	return 0;
}


///*
// * CSS-535 Lab 03: CUDA GEMV Implementation
// * Authors: Afrooz Rahmati & Tony Varela
// */
//
//
//#include <hip/hip_runtime.h>
//#include <>
//#include <hipblas.h> // as a benchmark 
//
//
//#include <random> // for random initialization
//#include <chrono> // timing
//#include <iostream> // for output 
//using namespace std;
//using namespace std::chrono;
//
////Matrix initialize with random values
///* void initialize_matrix(float *a, const int N, const int M) {
//        int i, j;
//        for(i=0; i<N; i++)
//            for(j=0; j<M; j++)
//                    a[i*M+j] = rand() % 4 + 1;
//} */
//
////Vector initialize with random values
///* void initialize_vector(float *a, const int N) {        
//        for(int i=0; i<N; i++)
//            a[i] = rand() % 4 + 1;
//} */
//
////GEMV naive Implementation
///* __global__ void multiplication(float *vec, float *mat, float *res, const int N, const int M){
//    
//    int tid= threadIdx.x+ blockIdx.x*blockDim.x;
//    float sum=0.0f;
//    
//    if(tid<M){
//        for(int i=0; i<N; i++)
//            sum += vec[i]*mat[(i*M)+tid];
//        res[tid]=sum;
//    }
//} */
//
///* 
//functionality: GEMV Implementation---Tiled version
//
//input parameters: 
//            vec         : the input vector 
//            mat         : the input matrix
//            res         : the result vector
//            N           : Matrix and Vector Size ( number of elements )
//            BLOCK_WIDTH : The GPU device block size
//consideration : the matrix size is square */
//
//#define BLOCK_SIZE  16  //should be change
//
//__global__ void mat_mul_tiled(float *vec, float *mat, float *res, const int N ){
//
//    __shared__  float As[BLOCK_SIZE][BLOCK_SIZE];
//    __shared__  float Bs[BLOCK_SIZE];
//	
//    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y; //the row index of As and Bs
//    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x; //the column index of As and Bs
//    float tmp = 0;
//    int idx;
//
//
//    for (int i = 0; i < gridDim.y; ++i) //initialize the ID index
//    {
//		
//        idx = row * N + i * BLOCK_SIZE + threadIdx.x;
//		
//        //if N is not divisible by block width
//        if (idx >= N*N)
//        {
//            As[threadIdx.y][threadIdx.x] = 0; 
//        }
//        else{
//            As[threadIdx.y][threadIdx.x] = mat[idx];   
//        }
//
//        idx = col + i * BLOCK_SIZE;
//
//
//        //if N is not divisible by block width
//        if(idx >= N)  
//        {
//            Bs[threadIdx.x] = 0;
//        }
//        else{
//            Bs[threadIdx.x] = vec[idx];  
//        }
//
//		
//
//        //Matrix and vectors should be loaded completely before any further process
//        __syncthreads();
//
//        //multiply sub matrices
//        for (int k = 0; k < BLOCK_SIZE; k++) 
//        {
//            tmp += As[threadIdx.y][k] * Bs[k];
//			
//        }
//
//        __syncthreads();
//    }
//
//    //write result back to global memory
//    if(row < N && col < N)
//    {
//        res[row] = tmp;  
//    }
//}
//
//
//
////printing the vector 
//void print_vector(float *a, const int N, char *d) {
//    int i;
//    for(i=0; i<N; i++)
//            printf("\n%s[%d]: %f",d, i, a[i]);
//    printf("\n");
//}
//
////printing the matrix values
//void print_matrix(float *a, const int N, const int M, char *d) {
//    int i, j;
//    for(i=0; i<N; i++){
//    printf("\n%s[%d]:", d, i);
//    for (j=0; j<M; j++)
//        printf("\t%6.4f", a[i*M+j]);
//    }
//    printf("\n");
//}
//
//
//
//// Credits to Brian Luger for the main structure of this program (just the way it is divided, I learned this from our time together on Lab 2)
//int main(int argc, char **argv) {
//	// TODO: create command line arguments to configure grid/block dimensions
//	// This program should only take in the M and N dimensions; within the program, we figure out the execution configurations ourselves
//
//	// cublas declarations
//	hipblasHandle_t cublas_handle;
//
//	// for now, let's put the matrix/vector dimensions in here as well
//	const size_t M{ 10000 };
//	const size_t N{ 10000 };
//	// yes, I know they're always going to be square, but I like separating M and N for my own understanding.
//	// TODO: consider experimenting with thrust device/host vectors as well
//
//	// seed RNG
//	std::default_random_engine dre;
//	dre.seed(3); // seeded for reproducibility
//	std::uniform_real_distribution<float> uniform_dist(-10, 10); // uniform distribution [-10, 10]
//
//	// allocate host memory
//	float *m{new float[M * N]};
//	float *v_in{new float[N]};
//	float *v_out_naive{new float[M]};
//	float *v_out_cublas{new float[M]};
//
//	// allocate device memory
//	float *d_m, *d_v_in, *d_v_out_naive, *d_v_out_cublas;
//	hipMalloc(reinterpret_cast<void**>(&d_m), sizeof(float) * M * N);
//	hipMalloc(reinterpret_cast<void**>(&d_v_in), sizeof(float) * N);
//	hipMalloc(reinterpret_cast<void**>(&d_v_out_naive), sizeof(float) * M);
//	hipMalloc(reinterpret_cast<void**>(&d_v_out_cublas), sizeof(float) * M);
//
//    //**************************These lines are for debugging purpose only************************
//
////   /* The elements of the first column */
////  m[0] = 1;
////  m[1] = 2;
//// 	m[2] = 3;
//// 	m[3] = 4;
////    /* The elements of the second column */
//	
//
//// 	m[N] = 1;
//// 	m[N + 1] = 1;
//// 	m[N + 2] = 2;
//// 	m[N + 3] = 1;
////    /* The elements of the third column */
//// 	m[N * 2] = 3;
//// 	m[N * 2 + 1] = 1;
//// 	m[N * 2 + 2] = 2;
//// 	m[N * 2 + 3] = 1;
////    /* The elements of the fourth column */
//// 	m[N * 3] = 5;
//// 	m[N * 3 + 1] = 4;
//// 	m[N * 3 + 2] = 7;
//// 	m[N * 3 + 3] = 3;
//
//
//
//
//
////    /* The elements of x and y */
////    v_in[0] = 1;
////    v_in[1] = 3;
////    v_in[2] = 1;
////    v_in[3] = 2;
//
/////////////////////////**************************************************
//
//	// initialize host array with random data
//
//	//for the matrix 
//	for (size_t i{0}; i < M; i++) for (size_t j{0}; j < N; j++) 
//      m[i * M + j] = uniform_dist(dre);
//
//    //print_matrix(m, N, M, "input Matrix");  
//	
//	//std::cout << "Printing Matrix:\n";
//	//for (size_t i{0}; i < M; i++) {
//	//	for (size_t j{0}; j < N; j++) {
//	//		std::cout << m[i * M + j] << ' ';
//	//	}
//	//	std::cout << '\n';
//	//}
//	// for the vector
//	for (size_t i{0}; i < N; i++) v_in[i] = uniform_dist(dre);
//	//std::cout << "Printing Input Vector:\n";
//	//for (size_t i{0}; i < N; i++) std::cout << v_in[i] << ' ';
//
//    //print_vector(v_in, N, "input vector");
//
//	std::cout << '\n';
//	// copy m and v_in into device memory, time it as well
//	auto d2h_start = std::chrono::high_resolution_clock::now();
//	hipMemcpy(d_m, m, sizeof(float) * M * N, hipMemcpyHostToDevice);
//	hipMemcpy(d_v_in, v_in, sizeof(float) * N, hipMemcpyHostToDevice);
//	auto d2h_end = std::chrono::high_resolution_clock::now();
//	auto d2h_duration = std::chrono::duration_cast<std::chrono::microseconds>(d2h_end - d2h_start).count();
//
//	// TODO: there are CUBLAS operations for getting/setting matrices/vectors between host/device; consider looking/timing these as well: https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf (pg.48-49)
//
//	// let's create the grid / block configuration, but just really simply.
//	
//    //*****************************************************************************************
//    /////////////////specific to part 2////////////////////////////////////////////////////////////////
//    //const unsigned int BLOCK_SIZE = 16; ///we need to change it
//    unsigned int gridrows =  (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
//
//    dim3 grid ( 1 , gridrows );  
//    dim3 block(BLOCK_SIZE,BLOCK_SIZE);
//    
//
//	std::cout << "STARTING NAIVE" << std::endl;
//	auto naive_exec_start = std::chrono::high_resolution_clock::now();
//    mat_mul_tiled<<<grid, block>>>( d_v_in, d_m, d_v_out_naive, M);
//   
//    //naive_gemv <<<grid, block >>>(d_m, d_v_in, d_v_out_naive, M, N);
//	hipDeviceSynchronize();
//	std::cout << "FINISHED NAIVE" << std::endl;
//	// since the kernels are executed asynchronously, need to sync so that we can get accurate timing
//	auto naive_exec_end = std::chrono::high_resolution_clock::now();
//	auto naive_exec_duration = std::chrono::duration_cast<std::chrono::microseconds>(naive_exec_end - naive_exec_start).
//		count();
//	
//
//    //print_vector(d_v_out_naive, M, "out vector");
//
//
//	// copy d_v_out_naive back into host
//	auto h2d_start = std::chrono::high_resolution_clock::now();
//	hipMemcpy(v_out_naive, d_v_out_naive, sizeof(float) * M, hipMemcpyDeviceToHost);
//	auto h2d_end = std::chrono::high_resolution_clock::now();
//	auto h2d_duration = std::chrono::duration_cast<std::chrono::microseconds>(h2d_end - h2d_start).count();
//
//	// get total inclusive time 
//	auto gpu_transfer_total_duration = h2d_duration + d2h_duration;
//	
//	// try timing cublas (not timing inclusive times, although I am copying back out to host as well)
//	hipblasCreate(&cublas_handle);
//	// hipblasSetMatrix(M, N, sizeof(float), m, M, )
//
//	const float a{1.0f};
//	const float b{0.0f};
//	auto cublas_exec_start = std::chrono::high_resolution_clock::now();
//	hipblasSgemv(cublas_handle, HIPBLAS_OP_T, N, M, &a, d_m, N, d_v_in, 1, &b, d_v_out_cublas, 1);
//
//
//	auto cublas_exec_end = std::chrono::high_resolution_clock::now();
//	auto cublas_exec_duration = std::chrono::duration_cast<std::chrono::microseconds>(
//		cublas_exec_end - cublas_exec_start).count();
//
//	// copy the cublas device vector back out to host
//	hipMemcpy(v_out_cublas, d_v_out_cublas, sizeof(float) * M, hipMemcpyDeviceToHost);
//
//	std::cout << "Comparing output vectors:\n";
//	float rse{ 0.0f };
//	for (size_t i{ 0 }; i < M; i++) rse += abs(v_out_naive[i] - v_out_cublas[i]);
//	std::cout << "ERROR: " << rse << std::endl;
//
/////////
//
////print_vector(d_v_out_cublas, M, "out vector");
//
//
//	// std::cout << "Naive: ";
//	// for (size_t i{ 0 }; i < M; i++) std::cout << v_out_naive[i] << ' ';
//	// std::cout << '\n';
//	
//	// std::cout << "cuBLAS: ";
//	// for (size_t i{0}; i < M; i++) std::cout << v_out_cublas[i] << ' ';
//	// std::cout << '\n';
//
//
//
//
//	std::cout <<
//		"Total Inclusive Time, Naive Execution Time, cuBLAS Execution Time, Naive Total Time, cuBLAS Total Time\n";
//	std::cout << gpu_transfer_total_duration << ", " << naive_exec_duration << ", " << cublas_exec_duration << ", " <<
//		naive_exec_duration +
//		gpu_transfer_total_duration << ", " << cublas_exec_duration + gpu_transfer_total_duration << '\n';
//
//	// clean up
//	hipblasDestroy(cublas_handle);
//
//	hipFree(d_v_out_cublas);
//	hipFree(d_v_out_naive);
//	hipFree(d_v_in);
//	hipFree(d_m);
//
//	delete[] v_out_cublas;
//	delete[] v_out_naive;
//	delete[] v_in;
//	delete[] m;
//
//	return 0;
//}
//
